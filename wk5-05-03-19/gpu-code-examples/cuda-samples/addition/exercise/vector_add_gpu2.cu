#include "hip/hip_runtime.h"
// SDSC Summer Institute 2018
// Andreas Goetz (agoetz@sdsc.edu)

// CUDA program to add two vectors in parallel on the GPU
// version 2:
// launch a fixed number of blocks and threads
//
// /* FIXME */ comments need modifications
//

#include<stdio.h>

// define vector length, number of blocks NBL and threads per block TPB
#define N (255*2047)
#define NBL 256
#define TPB 128

//
// CUDA device function that adds two integer vectors
//
__global__ void add(int *a, int *b, int *c, int n){

  /* FIXME 
     INSERT HERE CODE TO CALCULATE REQUIRED INDEX AND STRIDE
   */

  while (tid < n) {
    c[tid] = a[tid] + b[tid];
    tid += stride;
  }

}

//
// main program
//
int main(void){

  int *h_a, *h_b, *h_c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);
  int i, err;

  // allocate host memory
  h_a = (int *) malloc(size);
  h_b = (int *) malloc(size);
  h_c = (int *) malloc(size);

// allocate device memory
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // initialize vectors
  for (i=0; i<N; i++){
    h_a[i] = i+1;
    h_b[i] = i+1;
  }

  // copy input data to device
  hipMemcpy(/* FIXME */);
  hipMemcpy(/* FIXME */);

  // add vectors by launching a sufficient number of blocks of the add() kernel
  printf("\nLaunching vector addition kernel...\n");
  printf("Vector length     = %d\n",N);
  printf("Blocks            = %d\n",NBL);
  printf("Threads per block = %d\n",TPB);
  printf("Kernel copies     = %d\n",NBL*TPB);
  add<<</* FIXME */>>>(d_a, d_b, d_c, N);

  // copy results back to host
  hipMemcpy(/* FIXME */);

  // deallocate memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // check results
  err = 0;
  for (i=0; i<N; i++){
    if (h_c[i] != 2*(i+1)) err = 1;
  }
  if (err != 0){
    printf("\n Error, %d elements do not match!\n\n", err);
  } else {
    printf("\n Success! All elements match.\n\n");
  }

  // deallocate host memory
  free(h_a);
  free(h_b);
  free(h_c);

  return err;

}
