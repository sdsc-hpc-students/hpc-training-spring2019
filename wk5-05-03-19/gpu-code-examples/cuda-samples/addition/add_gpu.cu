// SDSC SCC Training - GPU Computing and Programming
// May 3, 2019
// Andreas Goetz (agoetz@sdsc.edu)

// CUDA program to add two integer numbers on the GPU
//


#include <hip/hip_runtime.h>
#include<stdio.h>

//
// CUDA device function that adds two integer numbers
//
__global__ void add(int *a, int *b, int *c){
  
  *c = *a + *b;           

}

//
// main program
//
int main(void) {

  int h_a, h_b, h_c;     // host copies
  int *d_a, *d_b, *d_c;  // device copies
  int size = sizeof(int);

  // allocate device memory
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // setup input data
  h_a = 5;
  h_b = 7;

  // copy input data to device
  hipMemcpy(d_a, &h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &h_b, size, hipMemcpyHostToDevice);

  // launch kernel
  add<<<1,1>>>(d_a, d_b, d_c);

  // copy results back to host
  hipMemcpy(&h_c, d_c, size, hipMemcpyDeviceToHost);

  // deallocate memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // print results
  printf("\n Addition on CPU: %d + %d = %d\n", h_a, h_b, h_a + h_b);
  printf("\n Addition on GPU: %d + %d = %d\n\n",h_a, h_b, h_c);

  return 0;

}
