// SDSC SCC Training - GPU Computing and Programming
// May 3, 2019
// Andreas Goetz (agoetz@sdsc.edu)

// Hello World Program in CUDA C
//
// Notice this is identical to standard C
// All that changes is the file extension .c -> .cu
// This code can be compiled with the NVIDIA nvcc CUDA compiler
//


#include <hip/hip_runtime.h>
#include<stdio.h>

int main(void) {

  printf("Hello World!\n");
  return 0;

}
