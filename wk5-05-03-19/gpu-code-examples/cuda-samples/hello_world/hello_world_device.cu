// SDSC SCC Training - GPU Computing and Programming
// May 3, 2019
// Andreas Goetz (agoetz@sdsc.edu)

// Hello World Program in CUDA C
//
// Contains a function that is executed on the device (GPU)
//


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void my_kernel(void){
}

int main(void) {

  my_kernel<<<1,1>>>();
  printf("Hello World!\n");
  return 0;

}
