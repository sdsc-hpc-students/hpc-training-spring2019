// SDSC Summer Institute 2018
// Andreas Goetz (agoetz@sdsc.edu)

// CUDA program that performs 1D stencil operation in parallel on the GPU
//
// /* FIXME */ COMMENTS REQUIRE ATTENTION


#include <hip/hip_runtime.h>
#include<stdio.h>

// define vector length, stencil radius, 
#define N (1024*1024*8l)
#define RADIUS 3
#define GRIDSIZE 128
#define BLOCKSIZE 256

// -------------------------------------------------------
// CUDA device function that performs 1D stencil operation
// -------------------------------------------------------
__global__ void stencil_1D(int *in, int *out, long dim){

  long gindex = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = gridDim.x * blockDim.x;

  // Go through all data
  // Step all threads in a block to avoid synchronization problem
  while ( gindex < (dim + blockDim.x) ) {

    /* FIXME - CAN WE USE SHARED MEMORY? */

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++) {
      if ( gindex + offset < dim && gindex + offset > -1)
	result += in[gindex + offset];
    }

    // Store the result
    if (gindex < dim)
      out[gindex] = result;

    // Update global index and quit if we are done
    gindex += stride;

    __syncthreads();

  }

}

// ------------
// main program
// ------------
int main(void){

  int *h_in, *h_out;
  int *d_in, *d_out;
  long size = N * sizeof(int);
  int i, j, ij, result, err;

  // allocate host memory
  h_in = new int[N];
  h_out = new int[N];

  // initialize vector
  for (i=0; i<N; i++){
    //    h_in[i] = i+1;
    h_in[i] = 1;
  }

  // allocate device memory
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);

  // copy input data to device
  hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

  // Apply stencil by launching a sufficient number of blocks
  printf("\n---------------------------\n");
  printf("Launching 1D stencil kernel\n");
  printf("---------------------------\n");
  printf("Vector length     = %ld (%ld MB)\n",N,N*4/1024/1024);
  printf("Stencil radius    = %d\n",RADIUS);
  printf("Blocks            = %d\n",GRIDSIZE);
  printf("Threads per block = %d\n",BLOCKSIZE);
  printf("Total threads     = %d\n",GRIDSIZE*BLOCKSIZE);

  stencil_1D<<<GRIDSIZE,BLOCKSIZE>>>(d_in, d_out, N);

  // copy results back to host
  hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

  // deallocate device memory
  hipFree(d_in);
  hipFree(d_out);

  // check results
  err = 0;
  for (i=0; i<N; i++){
    result = 0;
    for (j=-RADIUS; j<=RADIUS; j++){
      ij = i+j;
      if (ij>=0 && ij<N)
        result += h_in[ij];
    }
    if (h_out[i] != result) {
      err++;
      // printf("h_out[%d]=%d\n",i,h_out[i]);
    }
  }

  if (err != 0){
    printf("\n Error, %d elements do not match!\n\n", err);
  } else {
    printf("\n Success! All elements match CPU result.\n\n");
  }

  // deallocate host memory
  free(h_in);
  free(h_out);

  return 0;

}
