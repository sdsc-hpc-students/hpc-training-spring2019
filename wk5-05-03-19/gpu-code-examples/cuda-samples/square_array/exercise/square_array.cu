#include "hip/hip_runtime.h"
// SDSC Summer Institute 2018
// Andreas Goetz (agoetz@sdsc.edu)

// CUDA program to square matrix elements in parallel on the GPU
//
// /* FIXME */ COMMENTS NEED MODIFICATION
//

#include<stdio.h>

// define matrix size, number of blocks NBL and threads per block TPB
#define NROW 2048
#define NCOL 512
#define NBLX 32
#define NBLY 32
#define TPBX 16
#define TPBY 16

//
// CUDA device function that squares elements of a 2D array
//
__global__ void square(int *arr, int maxrow, int maxcol){

  int rowinit = /* FIXME */;
  int colinit = threadIdx.y + blockDim.y * blockIdx.y;
  int rowstride = /* FIXME */
  int colstride = /* FIXME */
  int pos;

  // operate on all submatrices
  for (int row = rowinit; row < maxrow; row += rowstride) {
    for (int col = colinit; col < maxcol; col += colstride) {
      pos = row*maxcol + col;
      arr[pos] *= arr[pos];
    }
  }

}

//
// main program
//
int main(void){

  int h_a[NROW][NCOL];
  int *d_a;
  int size = NROW * NCOL * sizeof(int);
  int i, j, err;

  // allocate device memory
  hipMalloc((void **)&d_a, size);

  // initialize matrix
  for (i=0; i<NROW; i++){
    for (j=0; j<NCOL; j++){
      h_a[i][j] = i+j;
      // printf("Element (%d,%d) = %d\n",i,j,h_a[i][j]);
    }
  }

  // copy input data to device
  hipMemcpy(/* FIXME */);

  // add vectors by launching a sufficient number of blocks of the add() kernel
  printf("\nLaunching kernel to square matrix elements...\n");
  printf("Matrix elements   = %d x %d = %d\n",NROW,NCOL,NROW*NCOL);
  printf("Blocks            = %d x %d = %d\n",NBLX,NBLY,NBLX*NBLY);
  printf("Threads per block = %d x %d = %d\n",TPBX,TPBY,TPBX*TPBY);
  printf("Kernel copies     = %d\n",NBLX*NBLY*TPBX*TPBY);
  square<<</* FIXME */>>>(d_a, NROW, NCOL);

  // copy results back to host
  hipMemcpy(/* FIXME */);

  // deallocate memory
  hipFree(d_a);

  // check results
  err = 0;
  for (i=0; i<NROW; i++){
    for (j=0; j<NCOL; j++){
      if (h_a[i][j] != (i+j)*(i+j)) err += 1;
      //printf("Element (%d,%d) = %d\n",i,j,h_a[i][j]);
    }
  }
  if (err != 0){
    printf("\n Error, %d elements do not match!\n\n", err);
  } else {
    printf("\n Success! All elements match.\n\n");
  }

  return 0;

}
